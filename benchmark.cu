#include "hip/hip_runtime.h"
#include "gnuplot_i.hpp" // GnuPlot
#include "benchmark.h"

void drawPlotForPrimalityTestAlgorithms(Gnuplot gnuplot);
void drawPlotForResearchOfPrimes(Gnuplot gnuplot);
void drawPlotForPrimesFactorisation(Gnuplot gnuplot);

/*using namespace boost;*/

/** \brief  Je suis une fonction qui vient enrouler les fonctions de créations de
 *          graphes de visualisation des performances des algorithmes.
 */
void generatePlots()
{
    Gnuplot gnuplot;

    drawPlotForPrimalityTestAlgorithms(gnuplot);
    //drawPlotForResearchOfPrimes(gnuplot);
    //drawPlotForPrimesFactorisation(gnuplot);
}

/** \brief  Je suis une fonction qui fait des mesures de temps pour analyser les performances
 *          des algorithmes de calcul de la primalité d'un nombre.
 *          (temps/s)
 *          ^
 *          |
 *          |
 *          |______________> (log2(N)) N e [[4;35]]
 *
 */
void drawPlotForPrimalityTestAlgorithms(Gnuplot gnuplot)
{
    vector<uint64_t> samples = generatePrimalityTestsSamples();
    vector<float> timeMeasurements = generatePrimalityTestsMeasurement(samples);
    for (uint64_t log2Samples = 4,
            i = 0;
            i < LOG2MAX_ISP-4;
            log2Samples++,
            i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        samples.at(i) = log2Samples;
    }

    gnuplot.reset_plot();
    cout << endl << endl << "*** Graphe pour le test de Primalité ***" << endl;
    gnuplot.set_grid();
    gnuplot.set_style("lines")
    .plot_xy(
            samples,
            timeMeasurements,
             "Mesure de temps (en ms) pour un nombre binaire de N bits"
             );
    gnuplot.savetofigure("PrimalityTestCPU.pdf");
}

/** \brief  Je suis une fonction qui fait des mesures de temps pour analyser les performances
 *          des algorithmes de recherches de nombres premiers.
 *          (temps/s)
 *          ^
 *          |
 *          |
 *          |________________> log2(N) [[4;18[ ~60 pour N = 17 sur Macbook Pro A1502
 */
void drawPlotForResearchOfPrimes(Gnuplot gnuplot)
{
    vector<uint64_t> limits = generateResearchOfPrimesLimits();
    vector<float> timeMeasurements = generateResearchOfPrimesMeasurement(limits);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ROP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        limits.at(i) = log2Samples;
    }

    gnuplot.reset_plot();
    cout << endl << endl << "*** Graphe pour la recherche de nombres premiers ***" << endl;
    gnuplot.set_grid();
    gnuplot.set_style("lines")
            .plot_xy(
                    limits,
                    timeMeasurements,
                    "Mesure de temps (en ms) pour une borne superieure N."
            );
    wait_for_key();
}

/** \brief  Je suis une fonction qui fait des mesures de temps pour analyser les performances
 *          des algorithmes de factorisation en nombres premiers.
 *          (temps/s)
 *          ^
 *          |
 *          |
 *          |___________________> N Un nombre entier
 */
void drawPlotForPrimesFactorisation(Gnuplot gnuplot)
{
    vector<uint64_t> samples = generatePrimeFactorisationSamples();
    vector<float> timeMeasurements = generatePrimeFactorisationMeasurement(samples);

    gnuplot.reset_plot();
    cout << endl << endl << "*** Graphe pour la Factorisation en Nombres Premiers ***" << endl;
    gnuplot.set_grid();
    gnuplot.set_style("lines")
            .plot_xy(
                    samples,
                    timeMeasurements,
                    "Mesure de temps (en ms) pour un entier N."
            );
    wait_for_key();
}

void wait_for_key()
{
 cout << endl << "Appuyez sur une touche pour continuer." << endl;
 std::cin.clear();
 std::cin.ignore(std::cin.rdbuf()->in_avail());
 std::cin.get();
}
