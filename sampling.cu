#include "hip/hip_runtime.h"
#include "sampling.h"

/** \brief  Je suis une foncton qui aggrège les mesures de
 *          nos tests de performances pour une création de
 *          graphes ultérieure avec GnuPlot.
 * @return boost::tuple<vector<float>,vector<int>> couple de (Tableau de Mesures de temps, Tableau de Logarithme en base 2 des échantillons).
 *
boost::tuple<vector<float>,vector<uint64_t>>
createPrimalityTestsDatas()
{
    vector<uint64_t> samples = generatePrimalityTestsSamples();
    vector<float> timeMeasurements = generatePrimalityTestsMeasurement(samples);

    for (uint64_t log2Samples = 4,
            i = 0;
            i < LOG2MAX_ISP-4;
            log2Samples++,
            i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        samples.at(i) = log2Samples;
    }

    return boost::tuple<vector<float>, vector<uint64_t>>{timeMeasurements,samples};
}
*/

void generateDataFilesCPU(){
	cout << " Lancement des tests de performances et génération des fichiers de données " << endl;
    	generateResearchOfPrimesDataFileCPU();
    	generatePrimalityTestDataFileCPU();
    	generatePrimeFactorisationDataFileCPU(); 
    	cout << " Fin des tests de performances, les fichiers des résultats sont dans data/" << endl << endl;
}

void generateResearchOfPrimesDataFileCPU(){
	cout << " Génération des données pour la recherche de nombres premiers sur le CPU " << endl;
    vector<uint64_t> limits = generateResearchOfPrimesLimits();
    vector<float> timeMeasurements = generateResearchOfPrimesMeasurement(limits);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ROP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        limits.at(i) = log2Samples;
    }

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/researchOfPrimesCPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < limits.size(); i++){
		datafile << limits.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
    	cout << " Fin Génération " << endl;
}

void generatePrimalityTestDataFileCPU(){
	cout << " Génération des données pour le test de primalité sur le CPU " << endl;
    vector<uint64_t> samples = generatePrimalityTestsSamples();
    vector<float> timeMeasurements = generatePrimalityTestsMeasurement(samples);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ISP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        samples.at(i) = log2Samples;
    }

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/primalityTestsDatasCPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < samples.size(); i++){
		datafile << samples.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
	cout << " Fin Génération. " << endl;

}

void generatePrimeFactorisationDataFileCPU(){
	cout << " Génération des données pour la factorisation sur le CPU " << endl;
    vector<uint64_t> samples = generatePrimeFactorisationSamples();
    vector<float> timeMeasurements = generatePrimeFactorisationMeasurement(samples);

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/factorisationDatasCPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < samples.size(); i++){
		datafile << samples.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
	cout << " Fin Génération. " << endl;
}

void generateDataFilesGPU(){
	cout << " Lancement des tests de performances et génération des fichiers de données " << endl;
    generateResearchOfPrimesDataFileGPU();
    generatePrimalityTestDataFileGPU();
    generatePrimeFactorisationDataFileGPU(); 
    	cout << " Fin des tests de performances, les fichiers des résultats sont dans data/" << endl << endl;
}

void generateResearchOfPrimesDataFileGPU(){
	cout << " Génération des données pour la recherche de nombres premiers sur le GPU " << endl;
    vector<uint64_t> limits = generateResearchOfPrimesLimits();
    vector<float> timeMeasurements = generateGPUResearchOfPrimesMeasurement(limits);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ROP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        limits.at(i) = log2Samples;
    }

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/researchOfPrimesGPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < limits.size(); i++){
		datafile << limits.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
    	cout << " Fin Génération " << endl;
}

void generatePrimalityTestDataFileGPU(){
	cout << " Génération des données pour le test de primalité sur le GPU " << endl;
    vector<uint64_t> samples = generatePrimalityTestsSamples();
    vector<float> timeMeasurements = generateGPUPrimalityTestsMeasurement(samples);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ISP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        samples.at(i) = log2Samples;
    }

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/primalityTestsDatasGPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < samples.size(); i++){
		datafile << samples.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
	cout << " Fin Génération. " << endl;

}

void generatePrimeFactorisationDataFileGPU(){
	cout << " Génération des données pour la factorisation sur le GPU " << endl;
    vector<uint64_t> samples = generatePrimeFactorisationSamples();
    vector<float> timeMeasurements = generateGPUPrimeFactorisationMeasurement(samples);

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/factorisationDatasGPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < samples.size(); i++){
		datafile << samples.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
	cout << " Fin Génération. " << endl;
}

/** \brief  Je suis une fonction qui génère des nombres
 *          avec une longeur allant de 4 à 35 bits.
 *  @return vector<uint64_t> Les échantillons.
 */
vector<uint64_t> generatePrimalityTestsSamples() {
    vector<uint64_t> res(0);

    for (   uint64_t currentSample = 0b1000,
            i = 4;
            i < LOG2MAX_ISP;
            currentSample <<= 1,
            i++
    ) {
        res.push_back(currentSample);
    }

    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
 vector<float> generatePrimalityTestsMeasurement(vector<uint64_t> samples){
     vector<float> res(0);
     for (int i = 0; i < samples.size(); i++){
         ChronoCPU *currentChrono = new ChronoCPU();
         currentChrono->start();
         isPrimeCPU_v0(samples.at(i));
         currentChrono->stop();
         res.push_back(currentChrono->elapsedTime());
         delete currentChrono;
     }
     return res;
 }


/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
 vector<float> generateGPUPrimalityTestsMeasurement(vector<uint64_t> samples){
     vector<float> res(0);
     for (int i = 0; i < samples.size(); i++){
	     //printf("%d round N = %ld\n", i, samples.at(i));
	 /* GPU Routine Alloc */
         uint64_t N = samples.at(i);
	 uint64_t sqrtN = sqrt(N) + 1;
	 uint64_t nombresDePossiblesPremiers = N-2;

	 uint64_t *possibles_premiers = (uint64_t *)malloc(sizeof(uint64_t) * nombresDePossiblesPremiers);
	 for (int i = 0, j = 2; j < N; possibles_premiers[i] =j,i++,j++);
	 unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));

	 uint64_t *dev_possibles_premiers;
	 hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * nombresDePossiblesPremiers);
	 unsigned int *dev_res_operations;
	 hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));
	 
	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	 /* END GPU ROUTINE ALLOC*/
	 ChronoCPU *currentChrono = new ChronoCPU();
        // Start chrono
	 currentChrono->start();
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN); 
	currentChrono->stop(); 
	// end chrono
	
	/* GPU Routine Dealloc */
	 hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);
	 /* GPU Routine Dealloc */
	res.push_back(currentChrono->elapsedTime()); //push time 
         delete currentChrono;
     }
     return res;
 }

/** \brief  Je suis une fonction qui aggrège les mesures de
*          nos tests de performances de la recherche de nombre premiers
*          pour une création de graphes ultérieure avec GnuPlot.
* @return boost::tuple<vector<float>,vector<int>> couple de (Tableau de Mesures de temps, Tableau de Logarithme en base 2 des échantillons).
*
boost::tuple<vector<float>,vector<uint64_t>>
createResearchOfPrimesDatas()
{
    vector<uint64_t> limits = generateResearchOfPrimesLimits();
    vector<float> timeMeasurements = generateResearchOfPrimesMeasurement(limits);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ROP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        limits.at(i) = log2Samples;
    }

    return boost::tuple<vector<float>, vector<uint64_t>>{timeMeasurements,limits};
}
*/

/** \brief  Je suis une fonction qui génère des puissances de 2.
 *  @return vector<uint64_t> Les échantillons.
 */
vector<uint64_t> generateResearchOfPrimesLimits() {
    vector<uint64_t> res(0);

    for (   uint64_t currentSample = 0b1000,
                    i = 4;
            i < LOG2MAX_ROP;
            currentSample <<= 1,
                    i++
            ) {
        res.push_back(currentSample);
    }

    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
vector<float> generateResearchOfPrimesMeasurement(vector<uint64_t> limits){
    vector<float> res(0);
    for (int i = 0; i < limits.size(); i++){
        ChronoCPU *currentChrono = new ChronoCPU();
        currentChrono->start();
        searchPrimesCPU_v0(limits.at(i));
        currentChrono->stop();
        res.push_back(currentChrono->elapsedTime());
        delete currentChrono;
    }
    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
vector<float> generateGPUResearchOfPrimesMeasurement(vector<uint64_t> limits){
    vector<float> res(0);
    for (int i = 0; i < limits.size(); i++){
	uint64_t borne_sup = limits.at(i);
	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); possibles_premiers[i] = i+2, i++);
	uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); square_roots[i] = sqrt(i+2), i++);
	uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); premiers[i] = 0, i++);
       
	uint64_t *dev_possibles_premiers;
	uint64_t *dev_square_roots;
	uint64_t *dev_premiers;

	hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(borne_sup-2));
	
	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	
	ChronoCPU *currentChrono = new ChronoCPU();
        currentChrono->start();
	searchPrimeGPU<<<GRIDDIM(borne_sup-2),BLOCKDIM,SIZEMEM>>>(
			dev_possibles_premiers, 
			dev_square_roots, 
			borne_sup, 
			dev_premiers);
        currentChrono->stop();
        res.push_back(currentChrono->elapsedTime());
        delete currentChrono;
    	hipFree(dev_possibles_premiers);
    	hipFree(dev_square_roots);
    	hipFree(dev_premiers);
    	free(possibles_premiers);
    	free(square_roots);
    	free(premiers);
    }
    return res;
}


/** \brief  Je suis une fonction qui aggrège les mesures de
*          nos tests de performances de la recherche de nombre premiers
*          pour une création de graphes ultérieure avec GnuPlot.
* @return boost::tuple<vector<float>,vector<int>> couple de (Tableau de Mesures de temps, Tableau de Logarithme en base 2 des échantillons).
*
boost::tuple<vector<float>,vector<uint64_t>>
createPrimeFactorisationDatas()
{
    vector<uint64_t> samples = generatePrimeFactorisationSamples();
    vector<float> timeMeasurements = generatePrimeFactorisationMeasurement(samples);

    return boost::tuple<vector<float>, vector<uint64_t>>{timeMeasurements,samples};
}
*/

/** \brief  Je suis une fonction qui génère des puissances de 2.
 *  @return vector<uint64_t> Les échantillons.
 */
vector<uint64_t> generatePrimeFactorisationSamples() {
    vector<uint64_t> res(0);
    int limit = 40000;
    double step = limit/25;

    for (
            int i = 10;
            i < limit;
            i+=step){
        res.push_back(i);
    }

    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
vector<float> generatePrimeFactorisationMeasurement(vector<uint64_t> samples){
    vector<float> res(0);
    for (int i = 0; i < samples.size(); i++){
        ChronoCPU *currentChrono = new ChronoCPU();
        vector<cell> factors(0);
        currentChrono->start();
        factoCPU(samples.at(i), &factors);
        currentChrono->stop();
        res.push_back(currentChrono->elapsedTime());
        delete currentChrono;
    }
    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
vector<float> generateGPUPrimeFactorisationMeasurement(vector<uint64_t> samples){
    vector<float> res(0);
    for (int i = 0; i < samples.size(); i++){
        uint64_t N=samples.at(i);
	ChronoCPU *currentChrono = new ChronoCPU();

	vector<uint64_t> premiers_packed = getPrimes(N);
	int taille = premiers_packed.size(); 
	uint64_t *primes = (uint64_t*)malloc(sizeof(uint64_t) * taille);
	for(int i = 0; i < taille; primes[i]=premiers_packed.at(i),i++);

	cell  *facteurs=(cell*)malloc(sizeof(cell)*taille);
	for(int i =0 ; i<taille; i++) {
			facteurs[i].base=primes[i];
			facteurs[i].expo=0;
	}

	uint64_t *dev_primes;
	cell *dev_facteurs;
	
	hipMalloc((void**)&dev_primes,sizeof(uint64_t)*taille);
        hipMalloc((void**)&dev_facteurs,sizeof(cell)*taille);
       
	hipMemcpy(dev_primes,primes,sizeof(uint64_t)*taille,hipMemcpyHostToDevice);
	hipMemcpy(dev_facteurs,facteurs,sizeof(cell)*taille,hipMemcpyHostToDevice);
	currentChrono->start();
        
	factGPU<<<GRIDDIM(taille),BLOCKDIM>>>(
			N,
			dev_primes,
			taille,
			dev_facteurs);
        currentChrono->stop();
        
	res.push_back(currentChrono->elapsedTime());
        delete currentChrono;
	hipFree(dev_primes);
	hipFree(dev_facteurs);
	free(primes);
	free(facteurs);	
    }
    return res;
}
