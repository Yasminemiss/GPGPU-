#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <cstdlib>
#include <sstream>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <chrono>

#include "primeGPGPU.hpp"

using namespace std;
using namespace std::chrono;

#define BLOCKDIM 256
#define SIZEMEM (BLOCKDIM * sizeof(unsigned int))

uint64_t TailleGrid(uint64_t X){
	return ((X+BLOCKDIM-1)/BLOCKDIM);
}


void Lancer_isPrime(uint64_t N){
		uint64_t sqrtN = sqrt(N) + 1;
		uint64_t nombresDePossiblesPremiers = N-2;

		uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
		for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
		unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * TailleGrid(sqrtN));
		for (int i = 0; i < TailleGrid(sqrtN); res_operations[i] = 1,i++);

		uint64_t *dev_possibles_premiers;
		hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));

		unsigned int *dev_res_operations;
		hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * TailleGrid(sqrtN));


		hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
	       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * TailleGrid(sqrtN), hipMemcpyHostToDevice);
		isPrimeGPU<<<TailleGrid(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
		hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * TailleGrid(sqrtN), hipMemcpyDeviceToHost);


			std::cout <<" N "<< N << " est premier ? " << res_operations[0] << '\n';
		hipFree(dev_possibles_premiers);
		hipFree(dev_res_operations);
		free(possibles_premiers);
		free(res_operations);
}

vector<uint64_t> Lancer_searchPrimes(uint64_t N){



	vector<uint64_t> premiers_packed(0);
		uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); possibles_premiers[i] = i+2, i++);
		uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); square_roots[i] = sqrt(i+2), i++);
		uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); premiers[i] = 0, i++);


		uint64_t *dev_possibles_premiers;
		uint64_t *dev_square_roots;
		uint64_t *dev_premiers;
		hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(N-2));
		hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(N-2));
		hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(N-2));


		hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		uint64_t tailleGrid = (((N-2)+BLOCKDIM-1)/BLOCKDIM);
		searchPrimeGPU<<<tailleGrid,BLOCKDIM,SIZEMEM>>>(
				dev_possibles_premiers,
				dev_square_roots,
				N,
				dev_premiers);

		hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(N-2), hipMemcpyDeviceToHost);



		int nombresDePremiers = 0;
		for(int i = 0; i < (N-2); i++){
			if (premiers[i] != 0)
				nombresDePremiers++;
		}

		for (int i = 0; i < nombresDePremiers; i++){
			int j = 0;

			while (premiers[j] == 0 && j < (N-2))
				j++;
			premiers_packed.push_back(j+2);
			premiers[j] = 0;
		}


		return premiers_packed;
}

void Lancer_facteurs(uint64_t N){

  	vector<uint64_t> premiers_packed = Lancer_searchPrimes(N);
		int taille = premiers_packed.size();
		uint64_t *primes = (uint64_t*)malloc(sizeof(uint64_t) * taille);
		for(int i = 0; i < taille; primes[i]=premiers_packed.at(i),i++);

		fact  *facteurs=(fact*)malloc(sizeof(fact)*taille);
		for(int i =0 ; i<taille; i++) {
				facteurs[i].base=primes[i];
				facteurs[i].expo=0;
		}

		uint64_t *dev_primes;
		fact *dev_facteurs;

		hipMalloc((void**)&dev_primes,sizeof(uint64_t)*taille);
	  hipMalloc((void**)&dev_facteurs,sizeof(fact)*taille);

		hipMemcpy(dev_primes,primes,sizeof(uint64_t)*taille,hipMemcpyHostToDevice);
		hipMemcpy(dev_facteurs,facteurs,sizeof(fact)*taille,hipMemcpyHostToDevice);


	  factGPU<<<TailleGrid(taille),BLOCKDIM>>>(
				N,
				dev_primes,
				taille,
				dev_facteurs);

	     	hipMemcpy(facteurs,dev_facteurs,sizeof(fact)*taille,hipMemcpyDeviceToHost);


	     vector<fact> resulat(0);
	    for(int i=0 ; i <taille;i++)
	    {

	          if(facteurs[i].expo!=0)
	         {
	            fact c;
	           c.base=facteurs[i].base;
						 std::cout << c.base <<" voir " << '\n';
	          c.expo=facteurs[i].expo;
	          resulat.push_back(c);
	          }
	   }

		string res = "Les Facteurs premiers :  \n ";
	 for(int i = 0 ; i < resulat.size(); i++)
	 {
			 string cell = to_string(resulat.at(i).base)+"^"+to_string(resulat.at(i).expo);
			 res+= (i==resulat.size()-1) ? ""+cell : cell+"*" ;
	 }
	 std::cout << res << '\n';

}

int main( int argc, char **argv ){
  uint64_t N =atol(argv[1]);
  auto start = high_resolution_clock::now();

  Lancer_isPrime(N);
  vector<uint64_t> premiers_packed=Lancer_searchPrimes(N);
	string printable =  "La liste des  premiers : \n " ;

		for(int i =0 ; i < premiers_packed.size() ; i++)
		{
				printable += "-" + std::to_string(premiers_packed.at(i)) + "-";
		}

		std::cout <<printable << '\n';
  Lancer_facteurs(N);
  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
 cout << "Time taken by GPU version : "
        << duration.count() << " microseconds" << endl;

   	return 0;
}
