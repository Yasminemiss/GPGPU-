#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <cstdlib>
#include <sstream>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <chrono>

using namespace std;
using namespace std::chrono;


void isPrime(){


	uint64_t N = 32;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
  hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrimeGPU<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);
}

vector<uint64_t> searchPrimes(uint64_t borne_sup){

	vector<uint64_t> premiers_packed(0);
	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); possibles_premiers[i] = i+2, i++);
	uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); square_roots[i] = sqrt(i+2), i++);
	uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); premiers[i] = 0, i++);


	uint64_t *dev_possibles_premiers;
	uint64_t *dev_square_roots;
	uint64_t *dev_premiers;
	hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(borne_sup-2));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	searchPrimeGPU<<<GRIDDIM(borne_sup-2),BLOCKDIM,SIZEMEM>>>(
			dev_possibles_premiers,
			dev_square_roots,
			borne_sup,
			dev_premiers);

	hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyDeviceToHost);

	int nombresDePremiers = 0;
	for(int i = 0; i < (borne_sup-2); i++){
		if (premiers[i] != 0)
			nombresDePremiers++;
	}

	for (int i = 0; i < nombresDePremiers; i++){
		int j = 0;

		while (premiers[j] == 0 && j < (borne_sup-2))
			j++;
		premiers_packed.push_back(j+2);
		premiers[j] = 0;
	}

	return premiers_packed;
}

void facteurs(){


        uint64_t N = 30);
      	vector<uint64_t> premiers_packed = searchPrimes(N);
      	int taille = premiers_packed.size();
      	uint64_t *primes = (uint64_t*)malloc(sizeof(uint64_t) * taille);
      	for(int i = 0; i < taille; primes[i]=premiers_packed.at(i),i++);

      	uint64_t  *facteurs=(uint64_t*)malloc(sizeof(uint64_t)*taille);
      	for(int i =0 ; i<taille; i++) {
      			facteurs[i][0]=primes[i];
      			facteurs[i][1]=0;
      	}

      	uint64_t *dev_primes;
      	uint64_t *dev_facteurs;

      	hipMalloc((void**)&dev_primes,sizeof(uint64_t)*taille);
        hipMalloc((void**)&dev_facteurs,sizeof(uint64_t)*taille);
      	hipMemcpy(dev_primes,primes,sizeof(uint64_t)*taille,hipMemcpyHostToDevice);
      	hipMemcpy(dev_facteurs,facteurs,sizeof(uint64_t)*taille,hipMemcpyHostToDevice);


        factGPU<<<GRIDDIM(taille),BLOCKDIM>>>(N,dev_primes,taille,dev_facteurs);

        hipMemcpy(facteurs,dev_facteurs,sizeof(uint64_t)*taille,hipMemcpyDeviceToHost);

          vector<uint64_t*> resulat(0);
          for(int i=0 ; i <taille;i++)
          {
                if(facteurs[i][1]!=0)
               {
                  uint64_t c[0];
                 c[0]=facteurs[i][1];
                c[1]=facteurs[i][1];
                resulat.push_back(c);
                }
         }
}

int main( int argc, char **argv ){
  uint64_t N =30;
  auto start = high_resolution_clock::now();

  isPrime();
  vector<uint64_t> v=searchPrimes(N);
  facteurs();
  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
 cout << "Time taken by function: "
        << duration.count() << " microseconds" << endl;

   	return 0;
}
