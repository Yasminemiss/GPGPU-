#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <cstdlib>
#include <sstream>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <chrono>

#include "primeGPGPU.hpp"

using namespace std;
using namespace std::chrono;


#define GRID(X) ((X+BLOCKDIM-1)/BLOCKDIM)
#define SIZEMEM (BLOCKDIM * sizeof(unsigned int))
#define BLOCKDIM 256

void isPrime(uint64_t N){
		uint64_t sqrtN = sqrt(N) + 1;
		uint64_t nombresDePossiblesPremiers = N-2;

		uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
		for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
		unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRID(sqrtN));
		for (int i = 0; i < GRID(sqrtN); res_operations[i] = 1,i++);

		uint64_t *dev_possibles_premiers;
		hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));

		unsigned int *dev_res_operations;
		hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRID(sqrtN));


		hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
	       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRID(sqrtN), hipMemcpyHostToDevice);
		isPrime<<<GRID(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
		hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRID(sqrtN), hipMemcpyDeviceToHost);


			std::cout <<" N "<< N << " est premier ? " << res_operations[0] << '\n';
		hipFree(dev_possibles_premiers);
		hipFree(dev_res_operations);
		free(possibles_premiers);
		free(res_operations);
}

vector<uint64_t> searchPrimes(uint64_t N){



	vector<uint64_t> premiers_packed(0);
		uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); possibles_premiers[i] = i+2, i++);
		uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); square_roots[i] = sqrt(i+2), i++);
		uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); premiers[i] = 0, i++);


		uint64_t *dev_possibles_premiers;
		uint64_t *dev_square_roots;
		uint64_t *dev_premiers;
		hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(N-2));
		hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(N-2));
		hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(N-2));


		hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		searchPrimeGPU<<<GRID(N-2),BLOCKDIM,SIZEMEM>>>(
				dev_possibles_premiers,
				dev_square_roots,
				N,
				dev_premiers);

		hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(N-2), hipMemcpyDeviceToHost);

		int nombresDePremiers = 0;
		for(int i = 0; i < (N-2); i++){
			if (premiers[i] != 0)
				nombresDePremiers++;
		}

		for (int i = 0; i < nombresDePremiers; i++){
			int j = 0;

			while (premiers[j] == 0 && j < (N-2))
				j++;
			premiers_packed.push_back(j+2);
			premiers[j] = 0;
		}

		return premiers_packed;
}

void facteurs(uint64_t N){

  	vector<uint64_t> premiers_packed = searchPrimes(N);
		int taille = premiers_packed.size();
		uint64_t *primes = (uint64_t*)malloc(sizeof(uint64_t) * taille);
		for(int i = 0; i < taille; primes[i]=premiers_packed.at(i),i++);

		fact  *facteurs=(fact*)malloc(sizeof(fact)*taille);
		for(int i =0 ; i<taille; i++) {
				facteurs[i].base=primes[i];
				facteurs[i].expo=0;
		}

		uint64_t *dev_primes;
		fact *dev_facteurs;

		hipMalloc((void**)&dev_primes,sizeof(uint64_t)*taille);
	  hipMalloc((void**)&dev_facteurs,sizeof(fact)*taille);

		hipMemcpy(dev_primes,primes,sizeof(uint64_t)*taille,hipMemcpyHostToDevice);
		hipMemcpy(dev_facteurs,facteurs,sizeof(fact)*taille,hipMemcpyHostToDevice);


	  factGPU<<<GRID(taille),BLOCKDIM>>>(
				N,
				dev_primes,
				taille,
				dev_facteurs);

	     	hipMemcpy(facteurs,dev_facteurs,sizeof(fact)*taille,hipMemcpyDeviceToHost);


	     vector<fact> resulat(0);
	    for(int i=0 ; i <taille;i++)
	    {

	          if(facteurs[i].expo!=0)
	         {
	            fact c;
	           c.base=facteurs[i].base;
	          c.expo=facteurs[i].expo;
	          resulat.push_back(c);
	          }
	   }

}

int main( int argc, char **argv ){
  uint64_t N =atol(argv[1]);
  auto start = high_resolution_clock::now();

  isPrime(N);
  vector<uint64_t> v=searchPrimes(N);
  facteurs(N);
  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
 cout << "Time taken by GPU version : "
        << duration.count() << " microseconds" << endl;

   	return 0;
}
