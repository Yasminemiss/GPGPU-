#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <cstdlib>
#include <sstream>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <chrono>

#include "primeGPGPU.hpp"

using namespace std;
using namespace std::chrono;

#define BLOCKDIM 256
#define SIZEMEM (BLOCKDIM * sizeof(unsigned int))


void Lancer_isPrime(uint64_t N){
		uint64_t sqrtN = sqrt(N) + 1;
		uint64_t nombresDePossiblesPremiers = N-2;
					uint64_t tailleGrid = (((sqrtN)+BLOCKDIM-1)/BLOCKDIM);

		uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
		for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
		unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * tailleGrid);
		for (int i = 0; i < TailleGrid(sqrtN); res_operations[i] = 1,i++);

		uint64_t *dev_possibles_premiers;
		hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));

		unsigned int *dev_res_operations;



		hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * TailleGrid(sqrtN));


		hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
	       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * TailleGrid(sqrtN), hipMemcpyHostToDevice);
		isPrimeGPU<<<tailleGrid,BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
		hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * TailleGrid(sqrtN), hipMemcpyDeviceToHost);


			std::cout <<" N "<< N << " est premier ? " << res_operations[0] << '\n';
		hipFree(dev_possibles_premiers);
		hipFree(dev_res_operations);
		free(possibles_premiers);
		free(res_operations);
}

vector<uint64_t> Lancer_searchPrimes(uint64_t N){



	vector<uint64_t> premiers_packed(0);
		uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); possibles_premiers[i] = i+2, i++);
		uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); square_roots[i] = sqrt(i+2), i++);
		uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(N-2));
		for(int i = 0; i < (N-2); premiers[i] = 0, i++);


		uint64_t *dev_possibles_premiers;
		uint64_t *dev_square_roots;
		uint64_t *dev_premiers;
		hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(N-2));
		hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(N-2));
		hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(N-2));


		hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(N-2), hipMemcpyHostToDevice);
		uint64_t tailleGrid = (((N-2)+BLOCKDIM-1)/BLOCKDIM);
		searchPrimeGPU<<<tailleGrid,BLOCKDIM,SIZEMEM>>>(
				dev_possibles_premiers,
				dev_square_roots,
				N,
				dev_premiers);

		hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(N-2), hipMemcpyDeviceToHost);



		int nombresDePremiers = 0;
		for(int i = 0; i < (N-2); i++){
			if (premiers[i] != 0)
				nombresDePremiers++;
		}

		for (int i = 0; i < nombresDePremiers; i++){
			int j = 0;

			while (premiers[j] == 0 && j < (N-2))
				j++;
			premiers_packed.push_back(j+2);
			premiers[j] = 0;
		}


		return premiers_packed;
}

void Lancer_facteurs(uint64_t N){

  	vector<uint64_t> premiers_packed = Lancer_searchPrimes(N);

		uint64_t *primes = (uint64_t*)malloc(sizeof(uint64_t) * premiers_packed.size());
		for(int i = 0; i < premiers_packed.size(); primes[i]=premiers_packed.at(i),i++);

		fact  *facteurs=(fact*)malloc(sizeof(fact)*premiers_packed.size());
		for(int i =0 ; i<premiers_packed.size(); i++) {
				facteurs[i].base=primes[i];
				facteurs[i].expo=0;
				std::cout << "/* les primes  */"<<facteurs[i].base <<"expo "<< facteurs[i].expo<< '\n';
		}

		uint64_t *dev_primes;
		fact *dev_facteurs;

		hipMalloc((void**)&dev_primes,sizeof(uint64_t)*premiers_packed.size());
	  hipMalloc((void**)&dev_facteurs,sizeof(fact)*premiers_packed.size());

		hipMemcpy(dev_primes,primes,sizeof(uint64_t)*premiers_packed.size(),hipMemcpyHostToDevice);
		hipMemcpy(dev_facteurs,facteurs,sizeof(fact)*premiers_packed.size(),hipMemcpyHostToDevice);

			uint64_t tailleGrid = (((premiers_packed.size())+BLOCKDIM-1)/BLOCKDIM);
	  factGPU<<<tailleGrid,BLOCKDIM>>>(
				N,
				dev_primes,
				taille,
				dev_facteurs);

	     	hipMemcpy(facteurs,dev_facteurs,sizeof(fact)*premiers_packed.size(),hipMemcpyDeviceToHost);


	     vector<fact> resulat(0);
	    for(int i=0 ; i <premiers_packed.size();i++)
	    {
	          if(facteurs[i].expo!=0)
	         {
	            fact c;
	           c.base=facteurs[i].base;
	          c.expo=facteurs[i].expo;
	          resulat.push_back(c);
	          }
	   }

}

int main( int argc, char **argv ){
  uint64_t N =atol(argv[1]);
  auto start = high_resolution_clock::now();

  Lancer_isPrime(N);
  vector<uint64_t> premiers_packed=Lancer_searchPrimes(N);
	string printable =  "La liste des  premiers : \n " ;

		for(int i =0 ; i < premiers_packed.size() ; i++)
		{
				printable += "-" + std::to_string(premiers_packed.at(i)) + "-";
		}

		std::cout <<printable << '\n';
  Lancer_facteurs(N);
  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
 cout << "Time taken by GPU version : "
        << duration.count() << " microseconds" << endl;

   	return 0;
}
