#include "hip/hip_runtime.h"
#include "TestPrimeBreakerGPU.hpp"

/**	\brief	Je suis une fonction qui lance les tests unitaires pour les 
 * 		calculs a effectuer sur le GPU.
 */
void launchUnitTestGPU(){
    cout << "============================================"	<< endl;
    cout << "         Lancement des tests unitaires.     " 	<< endl;
    cout << "============================================"	<< endl << endl;

    testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfPrimesBetween0and100AreComputedOnGPU();
    testIfNumberIsFactorized();
    
    cout << "============================================"	<< endl;
    cout << "    Tests unitaires éffectués avec succès.   " 	<< endl;
    cout << "============================================"	<< endl << endl;
}


/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT32_T) n'est pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre non premier assez large tenant sur un UINT32_T n'est pas reconnu comme tel." << std::endl;

	uint64_t N = UINT32_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));

	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre non premier n'a pas été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT32_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre premier assez large tenant sur un UINT32_T est reconnu comme tel." << std::endl;
    
	uint64_t N = UINT32_T_PRIME;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT64_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre premier tenant sur un UINT64_T est reconnu comme tel." << std::endl;
	
    	uint64_t N = UINT64_T_PRIME;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT64_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;

}

/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT64_T) n'est  pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre non premier tenant sur un UINT64_T n'est pas reconnu comme tel." << std::endl;
	
    	uint64_t N = UINT64_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT64_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);


    std::cout << "Le nombre non premier n'a pas été reconnu : succès " << std::endl
    << std::endl;
}

void testIfPrimesBetween0and100AreComputedOnGPU(){
    std::cout << "Tester la récupération des nombres premiers entre 0 et 100." << std::endl;

	vector<uint64_t> controlPrimeSet = getPrimesFrom0to100FromControlPrimeSetFile();

	uint64_t borne_sup = 100;
	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); possibles_premiers[i] = i+2, i++);
	uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); square_roots[i] = sqrt(i+2), i++);
	uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); premiers[i] = 0, i++);

	if (VERBOSE) {
		cout << "Afficher les données initialisées " << endl;
		cout << "possibles premiers ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << possibles_premiers[i] << "]";
		}
		cout << endl << endl;
		
		cout << "square_roots ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << square_roots[i] << "]";
		}
		cout << endl << endl;
		
		cout << "premiers ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << premiers[i] << "]";
		}
		cout << endl << endl;

		cout << "Fin affichage des données initialisées" << endl;
	}


	uint64_t *dev_possibles_premiers;
	uint64_t *dev_square_roots;
	uint64_t *dev_premiers;

	hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(borne_sup-2));
	
	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);

	searchPrimeGPU<<<GRIDDIM(borne_sup-2),BLOCKDIM,SIZEMEM>>>(
			dev_possibles_premiers, 
			dev_square_roots, 
			borne_sup, 
			dev_premiers);

	hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyDeviceToHost);

	if (VERBOSE) {
		cout << "Affichage du tableau premiers après calcul GPU" << endl;
		for (int i = 0; i < borne_sup-2; i++){
			cout << "[" << i+2 << "," << premiers[i] << "]";
		}
		cout << endl << "Fin Affichage" << endl << endl; 
	}

    // Début Assertions

	int nombresDePremiers = 0;
	for(int i = 0; i < (borne_sup-2); i++){
		if (premiers[i] != 0)
			nombresDePremiers++;
	}
	uint64_t premiers_packed[nombresDePremiers];
	for (int i = 0; i < nombresDePremiers; i++){
		int j = 0;

		while (premiers[j] == 0 && j < (borne_sup-2))
			j++;
		premiers_packed[i] = (j+2);
		premiers[j] = 0;
	}

	if (VERBOSE)

        {
		printf("Liste de nombres premiers récupérés du GPU : \n");
		for (int i = 0; i < nombresDePremiers; i++){
			printf("[%lld]", premiers_packed[i]);
		}

		printf("\n\nListe des nombres premiers récupérés du témoin : \n");
		for (int i = 0; i < controlPrimeSet.size(); i++){
			printf("[%lld]", controlPrimeSet.at(i));
		}
	}

    mAssert("controlPrimeSet.size() == nombresDePremiers",
            controlPrimeSet.size() == nombresDePremiers,
            string("La fonction ne renvoit pas le même nombre de nombres premiers que dans le groupe de controle.\n")
            + string("controlPrimeSet.size() = ") + std::to_string(controlPrimeSet.size()) +
            string("\nprimesNumberFrom0to100.size() = ") + std::to_string(nombresDePremiers)
            + string("\n")
    );

    for (int i = 0; i < controlPrimeSet.size(); i++){
        mAssert("controlPrimeSet.at(i) == primesNumberFrom0to100.at(1)",
                controlPrimeSet.at(i) == premiers_packed[i],
                ("On ne retrouve pas le " + std::to_string(i) + "ème nombre premier.")
                );
    }

    hipFree(dev_possibles_premiers);
    hipFree(dev_square_roots);
    hipFree(dev_premiers);
    free(possibles_premiers);
    free(square_roots);
    free(premiers);

    std::cout << "On retrouve bien tout les nombres premiers compris dans l'interval : Succès." << std::endl << std::endl;
}


void  testIfNumberIsFactorized()
{
    std::cout << "Tester la factorisation d'un nombre entier sur le GPU" << std::endl;
        uint64_t N=100;
	
	cell cinq;
	cinq.base=5;
	cinq.expo=2;

	cell deux;
	deux.base=2;
	deux.expo=2;

	vector<uint64_t> premiers_packed = getPrimes(100);
	int taille = premiers_packed.size(); 
	uint64_t *primes = (uint64_t*)malloc(sizeof(uint64_t) * taille);
	for(int i = 0; i < taille; primes[i]=premiers_packed.at(i),i++);

	cell  *facteurs=(cell*)malloc(sizeof(cell)*taille);
	for(int i =0 ; i<taille; i++) {
			facteurs[i].base=primes[i];
			facteurs[i].expo=0;
	}

	uint64_t *dev_primes;
	cell *dev_facteurs;
	
	hipMalloc((void**)&dev_primes,sizeof(uint64_t)*taille);
        hipMalloc((void**)&dev_facteurs,sizeof(cell)*taille);
       
	hipMemcpy(dev_primes,primes,sizeof(uint64_t)*taille,hipMemcpyHostToDevice);
	hipMemcpy(dev_facteurs,facteurs,sizeof(cell)*taille,hipMemcpyHostToDevice);
      
    	
        factGPU<<<GRIDDIM(taille),BLOCKDIM>>>(
			N,
			dev_primes,
			taille,
			dev_facteurs);
	
     	hipMemcpy(facteurs,dev_facteurs,sizeof(cell)*taille,hipMemcpyDeviceToHost);

	mAssert("facteurs[0].expo == deux.expo[0]",
		facteurs[0].expo == deux.expo,
		"La puissance de 2 n'est pas correcte pour N = 100 expo = " + std::to_string(facteurs[0].expo) + "\n"
		);

	mAssert("facteurs[2].expo == cinq.expo[0]",
		facteurs[2].expo == cinq.expo,
		"La puissance de 5 n'est pas correcte pour N = 100 expo = " + std::to_string(facteurs[2].expo) + "\n");
	hipFree(dev_primes);
	hipFree(dev_facteurs);
	free(primes);
	free(facteurs);	
	cout << "La factorisation a bien fonctionné : Succès "<<endl<<endl;
	
}
	
/* \brief	Je suis une fonction qui permet d'obtenir un tableau de
   		nombres premiers jusqu'à une certaine borne.
*/
vector<uint64_t> getPrimes(uint64_t borne_sup){
	vector<uint64_t> premiers_packed(0);
		
	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); possibles_premiers[i] = i+2, i++);
	uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); square_roots[i] = sqrt(i+2), i++);
	uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); premiers[i] = 0, i++);


	uint64_t *dev_possibles_premiers;
	uint64_t *dev_square_roots;
	uint64_t *dev_premiers;
	hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(borne_sup-2));
	
	
	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	searchPrimeGPU<<<GRIDDIM(borne_sup-2),BLOCKDIM,SIZEMEM>>>(
			dev_possibles_premiers, 
			dev_square_roots, 
			borne_sup, 
			dev_premiers);

	hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyDeviceToHost);

	int nombresDePremiers = 0;
	for(int i = 0; i < (borne_sup-2); i++){
		if (premiers[i] != 0)
			nombresDePremiers++;
	}

	for (int i = 0; i < nombresDePremiers; i++){
		int j = 0;

		while (premiers[j] == 0 && j < (borne_sup-2))
			j++;
		premiers_packed.push_back(j+2);
		premiers[j] = 0;
	}

	return premiers_packed;
}	



 void lancerFactorizedWithInputGPU(int argc,char **argv)
{

          if(argc < 2) printUsage(argv[0]);



          uint64_t N = atoll(argv[2]);
        
	vector<uint64_t> premiers_packed = getPrimes(N);
	int taille = premiers_packed.size(); 
	uint64_t *primes = (uint64_t*)malloc(sizeof(uint64_t) * taille);
	for(int i = 0; i < taille; primes[i]=premiers_packed.at(i),i++);

	cell  *facteurs=(cell*)malloc(sizeof(cell)*taille);
	for(int i =0 ; i<taille; i++) {
			facteurs[i].base=primes[i];
			facteurs[i].expo=0;
	}

	uint64_t *dev_primes;
	cell *dev_facteurs;
	
	hipMalloc((void**)&dev_primes,sizeof(uint64_t)*taille);
        hipMalloc((void**)&dev_facteurs,sizeof(cell)*taille);
       
	hipMemcpy(dev_primes,primes,sizeof(uint64_t)*taille,hipMemcpyHostToDevice);
	hipMemcpy(dev_facteurs,facteurs,sizeof(cell)*taille,hipMemcpyHostToDevice);
      
    	ChronoGPU chrGPU;
      chrGPU.start();
        factGPU<<<GRIDDIM(taille),BLOCKDIM>>>(
			N,
			dev_primes,
			taille,
			dev_facteurs);
	
     	hipMemcpy(facteurs,dev_facteurs,sizeof(cell)*taille,hipMemcpyDeviceToHost);
        chrGPU.stop();
       
     vector<cell> resulat(0);
    for(int i=0 ; i <taille;i++)
    {

          if(facteurs[i].expo!=0)
         {
            cell c;
           c.base=facteurs[i].base;
          c.expo=facteurs[i].expo;
          resulat.push_back(c);
          }

   }
          const float timeComputeGPUFact = chrGPU.elapsedTime();
           cout << " Temps de factorisation en nombre premier : " << timeComputeGPUFact << " ms "<<endl;
          cout << " Factorisation GPU " <<printFacteurs(resulat)<<endl;


}

void lancerIsPrimeWithInputGPU(int argc,char **argv)
{

    	uint64_t N = UINT64_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
         ChronoGPU chrGPU;
      chrGPU.start();	
isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
      chrGPU.stop();
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);
       const float timeGPU = chrGPU.elapsedTime();
        cout<< " Temps du test de primalite " << timeGPU << " ms "<<endl;
        cout << "Est Premier ? " <<res_operations[0] <<endl;
}



void  lancerSearchPrimesGPU(int argc,char **argv)
{
	

	uint64_t borne_sup = atoll(argv[2]);
	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); possibles_premiers[i] = i+2, i++);
	uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); square_roots[i] = sqrt(i+2), i++);
	uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); premiers[i] = 0, i++);

	


	uint64_t *dev_possibles_premiers;
	uint64_t *dev_square_roots;
	uint64_t *dev_premiers;

	hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(borne_sup-2));
	
	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);

 	hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);

        ChronoGPU chrGPU;
     chrGPU.start();
	searchPrimeGPU<<<GRIDDIM(borne_sup-2),BLOCKDIM,SIZEMEM>>>(
			dev_possibles_premiers, 
			dev_square_roots, 
			borne_sup, 
			dev_premiers);
   chrGPU.stop();
	hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyDeviceToHost);

   const float timeGPU = chrGPU.elapsedTime();
   cout << " Temps de recherche : " << timeGPU <<" ms" <<endl;


    
}
