#include "hip/hip_runtime.h"
#include "primeGPGPU.hpp"


__device__ void warpReduce(volatile unsigned int* cache, int tid)
{
	cache[tid]=umin( cache[tid], cache[tid + 32] );
	cache[tid]=umin( cache[tid], cache[tid + 16] );
	cache[tid]=umin( cache[tid], cache[tid + 8] );
	cache[tid]=umin( cache[tid], cache[tid + 4] );
	cache[tid]=umin( cache[tid], cache[tid + 2] );
	cache[tid]=umin( cache[tid], cache[tid + 1] );
}

__global__
void isPrimeGPU(
		uint64_t *Prime_PossiblE,
		unsigned int *resultat,
		uint64_t N,
		uint64_t sqrtN
		){
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	int initial_gid = gid;
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	extern __shared__ unsigned int Shared_memory[];
	Shared_memory[tid] = 1;
	while (gid < sqrtN){

		if(N%Prime_PossiblE[gid] !=0){
				Shared_memory[tid] = 1;
		}else{
				Shared_memory[tid] = 0;
		}
		__syncthreads();
		int i = blockDim.x/2;
		while (i >32) {

			if (tid < i) {
				Shared_memory[tid] = umin ( Shared_memory[tid], Shared_memory[tid+i] );
			}
			__syncthreads();
			i /= 2;
		}
    if(tid < 32) warpReduce(Shared_memory,tid);
		if (tid == 0) {
			resultat[bid] = Shared_memory[0];
		}
		gid += gridDim.x * blockDim.x;
	}
	if (initial_gid < ((sqrtN+blockDim.x-1)/blockDim.x))
		resultat[0] = ((resultat[0] != 0) && (resultat[initial_gid] != 0));
}


__global__ void searchPrimeGPU(
		uint64_t *Prime_PossiblE,
		uint64_t *carre,
		uint64_t limit,
		uint64_t *prime)
{
	int t_id = threadIdx.x + blockIdx.x * blockDim.x;

	while (t_id < limit-2) {
		if (t_id == 0) {prime[t_id] = 1; }

		 else{


			int resultat_size = ((carre[t_id]+blockDim.x-1)/blockDim.x)+1;
			unsigned int *resultat = (unsigned int*)malloc(sizeof(unsigned int)*resultat_size);

			int i = Prime_PossiblE[t_id]-1;
			int stopBoucle=0;
	    while(i >= 2 && stopBoucle==0)
	    {
	        if (floor(Prime_PossiblE[t_id]/i) == Prime_PossiblE[t_id]/i){
						resultat[0]=0;
						stopBoucle=1;
					}
	        i--;
	    }
			if(stopBoucle==1) 		resultat[0]=1;
			hipDeviceSynchronize();


			prime[t_id] = resultat[0];
			free(resultat);
		}
		t_id += gridDim.x * blockDim.x;
	}

}


__global__
void factGPU(
		uint64_t  N,
		uint64_t *res_primes,
               	int size,
		fact *res_facteurs
)
{
	int index_grid = threadIdx.x+blockIdx.x*blockDim.x;
	int tid = threadIdx.x;
  extern __shared__ unsigned int Shared_memory[];

	while(index_grid < size){


    Shared_memory[tid] = 0;
		uint64_t tmp = N;

		while(tmp%res_primes[index_grid]==0){
			Shared_memory[tid] += 1;
			tmp = tmp/res_primes[index_grid];
		}

		__syncthreads();

		if (tid == 0){
			int i = 0;
			while ( i < blockDim.x){
				if (Shared_memory[i]==1) {
					res_facteurs[i+blockIdx.x*blockDim.x].expo =	res_facteurs[i+blockIdx.x*blockDim.x].expo + Shared_memory[i];
					N = N- (res_facteurs[i+blockIdx.x*blockDim.x].base * Shared_memory[i]);
				}
				i++;
			}
		}
		__syncthreads();

    index_grid+=blockDim.x*gridDim.x;

    }
}
