#include "hip/hip_runtime.h"
#include "primeGPGPU.hpp"

__global__
void isPrimeGPU(
		uint64_t *possibles_premiers,
		unsigned int *res_operations,
		uint64_t N,
		uint64_t sqrtN
		){

	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	int initial_gid = gid;
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	extern __shared__ unsigned int cache[];

	cache[tid] = 1;
	while (gid < sqrtN){
		cache[tid] = (N%possibles_premiers[gid] != 0);

		__syncthreads();

		int offset = blockDim.x/2;
		while (offset != 0) {
			if (tid < offset) {
				cache[tid] = umin ( cache[tid], cache[tid+offset] );
			}
			__syncthreads();
			offset /= 2;
		}

		if (tid == 0) {
			res_operations[bid] = cache[0];
		}

		gid += gridDim.x * blockDim.x;
	}


	if (initial_gid < ((sqrtN+blockDim.x-1)/blockDim.x))
		res_operations[0] = ((res_operations[0] != 0) && (res_operations[initial_gid] != 0));

}


__global__ void searchPrimeGPU(
		uint64_t *possibles_premiers,
		uint64_t *square_roots,
		uint64_t borne_sup,
		uint64_t *premiers)
{
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	while (gid < borne_sup-2) {
		if (gid == 0) {
			premiers[gid] = 1;
		} else {
			int res_operations_size = ((square_roots[gid]+blockDim.x-1)/blockDim.x)+1;
			unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int)*res_operations_size);
			isPrime<<<gridDim.x,blockDim.x,blockDim.x*sizeof(unsigned int)>>>
				(possibles_premiers,
			 	res_operations,
			 	possibles_premiers[gid],
			 	square_roots[gid]
			 	);
			hipDeviceSynchronize();

			premiers[gid] = res_operations[0];
			free(res_operations);
		}
		gid += gridDim.x * blockDim.x;
	}

}


__global__
void factGPU(
		uint64_t  N,
		uint64_t *dev_primes,
               	int taille,
		fact *dev_facteurs
)
{
	int gid = threadIdx.x+blockIdx.x*blockDim.x;
	int tid = threadIdx.x;
        extern __shared__ unsigned int cache[];

	while(gid < taille)
       	{
        	cache[tid] = 0;
		uint64_t temp_N = N;

		while(temp_N%dev_primes[gid]==0)
                {
			cache[tid] += 1;
			temp_N /= dev_primes[gid];
		}

		__syncthreads();

		if (tid == 0){
			for (int i = 0; i < blockDim.x; i++){
				if (cache[i]) {
					dev_facteurs[i+blockIdx.x*blockDim.x].expo += cache[i];
					N -= (dev_facteurs[i+blockIdx.x*blockDim.x].base * cache[i]);
				}
			}
		}
		__syncthreads();

            gid+=blockDim.x*gridDim.x;
        }
}
