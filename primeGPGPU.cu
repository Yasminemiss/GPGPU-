#include "hip/hip_runtime.h"
#include "primeGPGPU.hpp"


__device__ void warpReduce(volatile unsigned int* cache, int tid)
{
	cache[tid]=umin( cache[tid], cache[tid + 32] );
	cache[tid]=umin( cache[tid], cache[tid + 16] );
	cache[tid]=umin( cache[tid], cache[tid + 8] );
	cache[tid]=umin( cache[tid], cache[tid + 4] );
	cache[tid]=umin( cache[tid], cache[tid + 2] );
	cache[tid]=umin( cache[tid], cache[tid + 1] );
}

__global__
void isPrimeGPU(
		uint64_t *Prime_PossiblE,
		unsigned int *resultat,
		uint64_t N,
		uint64_t sqrtN
		){

	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	int initial_gid = gid;
	int bid = blockIdx.x;
	int tid = threadIdx.x;


	extern __shared__ unsigned int Shared_memory;

	Shared_memory[tid] = 1;
	while (gid < sqrtN){

		if(N%Prime_PossiblE[gid] !=0){
				Shared_memory[tid] = 1;
		}else{
				Shared_memory[tid] = 0;
		}

		__syncthreads();



		int i = blockDim.x/2;
		while (i >32) {

			if (tid < i) {
				Shared_memory[tid] = umin ( Shared_memory[tid], Shared_memory[tid+i] );
			}
			__syncthreads();
			i /= 2;
		}

    if(tid < 32) warpReduce(Shared_memory,tid);

		if (tid == 0) {
			resultat[bid] = Shared_memory[0];
		}

		gid += gridDim.x * blockDim.x;

	}


	if (initial_gid < ((sqrtN+blockDim.x-1)/blockDim.x))
		resultat[0] = ((resultat[0] != 0) && (resultat[initial_gid] != 0));

}


__global__ void searchPrimeGPU(
		uint64_t *Prime_PossiblE,
		uint64_t *square_roots,
		uint64_t borne_sup,
		uint64_t *premiers)
{
	/*int gid = threadIdx.x + blockIdx.x * blockDim.x;
	while (gid < borne_sup-2) {
		if (gid == 0) {
			premiers[gid] = 1;
		} else {
			int resultat_size = ((square_roots[gid]+blockDim.x-1)/blockDim.x)+1;
			unsigned int *resultat = (unsigned int*)malloc(sizeof(unsigned int)*resultat_size);

			isPrimeGPU<<<gridDim.x,blockDim.x,blockDim.x*sizeof(unsigned int)>>>
				(Prime_PossiblE,
			 	resultat,
			 	Prime_PossiblE[gid],
			 	square_roots[gid]
			 	);
			hipDeviceSynchronize();

			premiers[gid] = resultat[0];
			free(resultat);
		}
		gid += gridDim.x * blockDim.x;
	}*/

}


__global__
void factGPU(
		uint64_t  N,
		uint64_t *dev_primes,
               	int taille,
		fact *dev_facteurs
)
{
	int gid = threadIdx.x+blockIdx.x*blockDim.x;
	int tid = threadIdx.x;
        extern __shared__ unsigned int Shared_memory[];

	while(gid < taille)
       	{
        	Shared_memory[tid] = 0;
		uint64_t temp_N = N;

		while(temp_N%dev_primes[gid]==0)
                {
			Shared_memory[tid] += 1;
			temp_N /= dev_primes[gid];
		}

		__syncthreads();

		if (tid == 0){
			for (int i = 0; i < blockDim.x; i++){
				if (Shared_memory[i]) {
					dev_facteurs[i+blockIdx.x*blockDim.x].expo += Shared_memory[i];
					N -= (dev_facteurs[i+blockIdx.x*blockDim.x].base * Shared_memory[i]);
				}
			}
		}
		__syncthreads();

            gid+=blockDim.x*gridDim.x;
        }
}
