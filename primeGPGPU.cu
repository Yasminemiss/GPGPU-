#include "hip/hip_runtime.h"
#include "primeGPGPU.hpp"


__global__
void isPrimeGPU(
	uint64_t N_square,
	uint64_t N,
		uint64_t *p,
		unsigned int *r
		){

	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	int initial_gid = gid;
	int tid = threadIdx.x;
	extern __shared__ unsigned int cache[];

	cache[tid] = 1;
	while (gid < N_square){
		cache[tid] = (N%p[gid] != 0);

		__syncthreads();

		int offset = blockDim.x/2;
		while (offset != 0) {
			if (tid < offset) {
				cache[tid] = umin( cache[tid], cache[tid+offset] );
			}
			__syncthreads();
			offset /= 2;
		}

		if (tid == 0) {
			r[blockIdx.x] = cache[0];
		}

		gid += gridDim.x * blockDim.x;
	}


	if (initial_gid < ((N_square+blockDim.x-1)/blockDim.x))
	{
			r[0] = ((r[0] != 0) && (r[initial_gid] != 0));
	}


}


__global__ void searchPrimeGPU(
		uint64_t *possibles_premiers,
		uint64_t *square_roots,
		uint64_t borne_sup,
		uint64_t *premiers)
{
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	while (gid < borne_sup-2) {
		if (gid == 0) {
			premiers[gid] = 1;
		} else {
			int res_operations_size = ((square_roots[gid]+blockDim.x-1)/blockDim.x)+1;
			unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int)*res_operations_size);
			isPrime<<<gridDim.x,blockDim.x,blockDim.x*sizeof(unsigned int)>>>
				(possibles_premiers,
			 	res_operations,
			 	possibles_premiers[gid],
			 	square_roots[gid]
			 	);
			hipDeviceSynchronize();

			premiers[gid] = res_operations[0];
			free(res_operations);
		}
		gid += gridDim.x * blockDim.x;
	}

}

__global__
void factGPU(
		uint64_t  N,
		uint64_t *dev_primes,
               	int taille,
		uint64_t *dev_facteurs
)
{
	int gid = threadIdx.x+blockIdx.x*blockDim.x;
	int tid = threadIdx.x;
  extern __shared__ unsigned int cache[];
	while(gid < taille)
       	{
        	cache[tid] = 0;
		uint64_t temp_N = N;

		while(temp_N%dev_primes[gid]==0)
                {
			cache[tid] += 1;
			temp_N /= dev_primes[gid];
		}
		__syncthreads();
		if (tid == 0){
			for (int i = 0; i < blockDim.x; i++){
				if (cache[i]) {
					dev_facteurs[i+blockIdx.x*blockDim.x][1] += cache[i];
					N -= (dev_facteurs[i+blockIdx.x*blockDim.x][0] * cache[i]);
				}
			}
		}
		__syncthreads();
            gid+=blockDim.x*gridDim.x;
        }
}
