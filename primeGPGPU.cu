#include "hip/hip_runtime.h"
#include "primeGPGPU.hpp"


__device__ void warpReduce(volatile unsigned int* cache, int tid)
{
	cache[tid]=umin( cache[tid], cache[tid + 32] );
	cache[tid]=umin( cache[tid], cache[tid + 16] );
	cache[tid]=umin( cache[tid], cache[tid + 8] );
	cache[tid]=umin( cache[tid], cache[tid + 4] );
	cache[tid]=umin( cache[tid], cache[tid + 2] );
	cache[tid]=umin( cache[tid], cache[tid + 1] );
}

__global__
void isPrimeGPU(
		uint64_t *Prime_PossiblE,
		unsigned int *resultat,
		uint64_t N,
		uint64_t sqrtN
		){
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	int initial_gid = gid;
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	extern __shared__ unsigned int Shared_memory[];
	Shared_memory[tid] = 1;
	while (gid < sqrtN){

		if(N%Prime_PossiblE[gid] !=0){
				Shared_memory[tid] = 1;
		}else{
				Shared_memory[tid] = 0;
		}
		__syncthreads();
		int i = blockDim.x/2;
		while (i >32) {

			if (tid < i) {
				Shared_memory[tid] = umin ( Shared_memory[tid], Shared_memory[tid+i] );
			}
			__syncthreads();
			i /= 2;
		}
    if(tid < 32) warpReduce(Shared_memory,tid);
		if (tid == 0) {
			resultat[bid] = Shared_memory[0];
		}
		gid += gridDim.x * blockDim.x;
	}
	if (initial_gid < ((sqrtN+blockDim.x-1)/blockDim.x))
		resultat[0] = ((resultat[0] != 0) && (resultat[initial_gid] != 0));
}


__global__
void factGPU(
		uint64_t  N,
		uint64_t *res_primes,
               	int size,
		fact *res_facteurs
)
{
	int index_grid = threadIdx.x+blockIdx.x*blockDim.x;
	int tid = threadIdx.x;
  extern __shared__ unsigned int Shared_memory[];

	while(index_grid < size){


    Shared_memory[tid] = 0;
		uint64_t tmp = N;
		while(tmp%res_primes[index_grid]==0){
			Shared_memory[tid] += 1;
			tmp = tmp/res_primes[index_grid];
		}
		__syncthreads();

		if (tid == 0){
		//	int i = 0;
		//	while ( i < blockDim.x){
		for (int i = 0; i < blockDim.x; i++){
				if (Shared_memory[i]) {
					//res_facteurs[i+blockIdx.x*blockDim.x].expo =	res_facteurs[i+blockIdx.x*blockDim.x].expo + Shared_memory[i];
					//N = N- (res_facteurs[i+blockIdx.x*blockDim.x].base * Shared_memory[i]);


					res_facteurs[i+blockIdx.x*blockDim.x].expo += Shared_memory[i];
					N -=  (res_facteurs[i+blockIdx.x*blockDim.x].base * Shared_memory[i]);
				}
				//i++;
			}
		}
		__syncthreads();

    index_grid+=blockDim.x*gridDim.x;

    }
}



__global__ void searchPrimeGPU(
		uint64_t *Prime_PossiblE,
		uint64_t *carre,
		uint64_t limit,
		uint64_t *res_premes)
{
	int t_id = threadIdx.x + blockIdx.x * blockDim.x;
	while (t_id < limit-2) {
		if (t_id == 0) {
			res_premes[t_id] = 1;
		} else {
			int res_operations_size = ((carre[t_id]+blockDim.x-1)/blockDim.x)+1;
			unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int)*res_operations_size);


			isPrimeGPU<<<gridDim.x,blockDim.x,blockDim.x*sizeof(unsigned int)>>>
				(Prime_PossiblE,
			 	res_operations,
			 	Prime_PossiblE[t_id],
			 	carre[t_id]
			 	);
			hipDeviceSynchronize();

			res_premes[t_id] = res_operations[0];
			free(res_operations);
		}
		t_id += gridDim.x * blockDim.x;
	}

}
