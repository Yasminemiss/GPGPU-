#include "hip/hip_runtime.h"
#include "primeGPGPU.hpp"


__device__ void warpReduce(volatile unsigned int* cache, int tid)
{
	cache[tid]=umin( cache[tid], cache[tid + 32] );
	cache[tid]=umin( cache[tid], cache[tid + 16] );
	cache[tid]=umin( cache[tid], cache[tid + 8] );
	cache[tid]=umin( cache[tid], cache[tid + 4] );
	cache[tid]=umin( cache[tid], cache[tid + 2] );
	cache[tid]=umin( cache[tid], cache[tid + 1] );
}

__global__
void isPrimeGPU(
		uint64_t *Prime_PossiblE,
		unsigned int *resultat,
		uint64_t N,
		uint64_t sqrtN
		){
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	int initial_gid = gid;
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	extern __shared__ unsigned int Shared_memory[];
	Shared_memory[tid] = 1;
	while (gid < sqrtN){

		if(N%Prime_PossiblE[gid] !=0){
				Shared_memory[tid] = 1;
		}else{
				Shared_memory[tid] = 0;
		}
		__syncthreads();
		int i = blockDim.x/2;
		while (i >32) {

			if (tid < i) {
				Shared_memory[tid] = umin ( Shared_memory[tid], Shared_memory[tid+i] );
			}
			__syncthreads();
			i /= 2;
		}
    if(tid < 32) warpReduce(Shared_memory,tid);
		if (tid == 0) {
			resultat[bid] = Shared_memory[0];
		}
		gid += gridDim.x * blockDim.x;
	}
	if (initial_gid < ((sqrtN+blockDim.x-1)/blockDim.x))
		resultat[0] = ((resultat[0] != 0) && (resultat[initial_gid] != 0));
}

__global__
void factGPU(
		uint64_t  N,
		uint64_t *dev_primes,
               	int taille,
		cell *dev_facteurs
)
{
	int gid = threadIdx.x+blockIdx.x*blockDim.x;
	int tid = threadIdx.x;
        extern __shared__ unsigned int cache[];

	while(gid < taille)
       	{
        	cache[tid] = 0;
		uint64_t temp_N = N;

		while(temp_N%dev_primes[gid]==0)
                {
			cache[tid] += 1;
			temp_N /= dev_primes[gid];
		}

		__syncthreads();

		if (tid == 0){
			for (int i = 0; i < blockDim.x; i++){
				if (cache[i]) {
					dev_facteurs[i+blockIdx.x*blockDim.x].expo += cache[i];
					N -= (dev_facteurs[i+blockIdx.x*blockDim.x].base * cache[i]);
				}
			}
		}
		__syncthreads();

            gid+=blockDim.x*gridDim.x;
        }
}


__global__ void searchPrimeGPU(
		uint64_t *Prime_PossiblE,
		uint64_t *carre,
		uint64_t limit,
		uint64_t *res_premes)
{
	int t_id = threadIdx.x + blockIdx.x * blockDim.x;
	while (t_id < limit-2) {
		if (t_id == 0) {
			res_premes[t_id] = 1;
		} else {
			int res_operations_size = ((carre[t_id]+blockDim.x-1)/blockDim.x)+1;
			unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int)*res_operations_size);


			isPrimeGPU<<<gridDim.x,blockDim.x,blockDim.x*sizeof(unsigned int)>>>
				(Prime_PossiblE,
			 	res_operations,
			 	Prime_PossiblE[t_id],
			 	carre[t_id]
			 	);
			hipDeviceSynchronize();

			res_premes[t_id] = res_operations[0];
			free(res_operations);
		}
		t_id += gridDim.x * blockDim.x;
	}

}
